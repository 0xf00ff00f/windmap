#include "hip/hip_runtime.h"
#include "simulator.h"

#include <glm/gtc/constants.hpp>

#include <algorithm>
#include <random>
#include <cassert>
#include <cstdio>
#include <cstdlib>

// latitude: n/s, -pi/2 to pi/2
// longitude: e/w, -pi to pi
__device__ glm::vec3 latLonToCartesian(float lat, float lon)
{
    const auto r = glm::cos(lat);
    const auto x = r * glm::cos(lon);
    const auto y = r * glm::sin(lon);
    const auto z = glm::sin(lat);
    return {x, y, z};
}

__device__ glm::vec2 cartesianToLatLon(const glm::vec3 &position)
{
    float lon = glm::atan(position.y, position.x);
    float lat = glm::asin(position.z);
    return {lat, lon};
}

__global__ void updateParticle(Particle *particles, int particleCount, glm::vec2 *windMap, int windMapWidth,
                               int windMapHeight)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < particleCount; i += stride)
    {
        auto &particle = particles[i];

        const auto lat = particle.position.x;
        const auto lon = particle.position.y;
        auto position = latLonToCartesian(lat, lon);

        const auto windSpeed = [=] {
            const auto x =
                static_cast<int>(((lon + glm::pi<float>()) / (2.0f * glm::pi<float>())) * windMapWidth) % windMapWidth;
            const auto y =
                windMapHeight - 1 -
                static_cast<int>(((lat + glm::half_pi<float>()) / glm::pi<float>()) * windMapHeight) % windMapHeight;
            return windMap[y * windMapWidth + x];
        }();
        auto speed = particle.speed + 0.02f * windSpeed;

        // XXX check this
        auto n = glm::normalize(position);
        auto u = glm::cross(n, glm::vec3(0, 0, 1));
        auto v = glm::cross(n, u);
        position += v * speed.y + u * speed.x;
        position = glm::normalize(position);

        particle.position = cartesianToLatLon(position);

        particle.history[particle.historySize % Particle::MaxHistorySize] = particle.position;
        particle.historySize++;
    }
}

void Simulator::update()
{
    constexpr auto BlockSize = 256;
    constexpr auto NumBlocks = (ParticleCount + BlockSize - 1) / BlockSize;
    updateParticle<<<NumBlocks, BlockSize>>>(m_particles, ParticleCount, m_windMap, m_windMapWidth, m_windMapHeight);
    hipDeviceSynchronize();
}

Simulator::Simulator(const glm::vec2 *windMap, int windMapWidth, int windMapHeight)
    : m_windMapWidth(windMapWidth)
    , m_windMapHeight(windMapHeight)
{
    // initialize particles
    auto rv = hipMallocManaged(&m_particles, ParticleCount * sizeof(Particle));
    if (rv != hipSuccess)
    {
        fprintf(stderr, "hipMallocManaged failed (%d): %s\n", rv, hipGetErrorString(rv));
        abort();
    }

    std::mt19937 eng;
    std::uniform_real_distribution<> dist(0, 1);
    std::for_each(m_particles, m_particles + ParticleCount, [&](Particle &particle) {
        const auto lat = dist(eng) * glm::pi<float>() - glm::half_pi<float>();
        const auto lon = dist(eng) * 2.0f * glm::pi<float>() - glm::pi<float>();
        particle.position = glm::vec2(lat, lon);
        particle.speed = glm::vec2(0, 0);
    });

    // initialize wind map
    hipMallocManaged(&m_windMap, windMapWidth * windMapHeight * sizeof(glm::vec2));
    std::copy(windMap, windMap + windMapWidth * windMapHeight, m_windMap);
}

Simulator::~Simulator()
{
    hipFree(m_windMap);
    hipFree(m_particles);
}
